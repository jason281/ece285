#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <iostream>

/*
 * A simple introduction to programming in CUDA. This program prints "Hello
 * World from GPU! from 10 CUDA threads running on the GPU.
 */

__global__ void helloFromGPU()
{
    printf("Hello World from GPU! 1\n");
}

int main(int argc, char **argv)
{
    helloFromGPU<<<1, 10>>>();
	printf("Hello World from CPU!\n");
    CHECK(hipDeviceReset());
	char c;
	std::cin>>c;
    return 0;
}


