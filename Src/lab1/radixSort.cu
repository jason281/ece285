#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>

/*
Project: Lab1 Radix Sort
Class: UCSD ECE 285
Year: Spring 2018
Name: Yen Chang
PID: A53243772
*/

const int particles_num = 32;
const int cells_num = 16;

__global__ void rand_alloc(unsigned int* keys, unsigned int* value) {
	unsigned int idx = threadIdx.x + threadIdx.y*gridDim.x + blockIdx.x*gridDim.x*gridDim.y;
	hiprandState_t state;
	hiprand_init(0, 0, 0, &state);

	if (idx < particles_num) {
		keys[idx] = hiprand(&state) % cells_num;
		value[idx] = idx;
	}
}

__global__ void radix_sort(unsigned int* keys, unsigned int* value) {
	unsigned int idx = threadIdx.x + threadIdx.y*gridDim.x + blockIdx.x*gridDim.x*gridDim.y;
};

__global__ void particle_count(unsigned int* keys, unsigned int* value) {
	unsigned int idx = threadIdx.x + threadIdx.y*gridDim.x + blockIdx.x*gridDim.x*gridDim.y;
	//std::cout << keys[idx] << '\t' << value[idx] << std::endl;
}

int main() {
	unsigned int keys_mem_size = sizeof(unsigned int)*particles_num;
	unsigned int value_mem_size = sizeof(unsigned int)*particles_num;

	unsigned int *d_keys, *d_value;
	hipMalloc((void**)d_keys,  keys_mem_size);
	hipMalloc((void**)d_value, value_mem_size);

	dim3 threads(particles_num / 32, particles_num / 1024);
	dim3 blocks(particles_num / 2014 / 32);

	rand_alloc <<<blocks, threads >>> (d_keys, d_value);
	//radix_sort <<<blocks, threads >>> (d_keys, d_value);
	//particle_count <<<blocks, threads >>> (d_keys, d_value);

	unsigned int *h_keys, *h_value;
	hipMemcpy(h_keys, d_keys, sizeof(d_keys), hipMemcpyDeviceToHost);
	hipMemcpy(h_value, d_value, sizeof(d_value), hipMemcpyDeviceToHost);
	for(int i=0;i<particles_num;i++)
		std::cout << h_keys[i] << '\t' << h_value[i] << std::endl;


	CHECK(hipDeviceReset());
	char c;
	std::cin >> c;
	return 0;
}